#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

__global__ void MyKernelFunc(int* d_input, const int* d_range, const int* d_rows, const int* d_cols, int numMatrices){
	int k = blockIdx.x;
    if (k >= numMatrices) return;
	int maxV = d_range[k];
    int rows = d_rows[k];
    int cols = d_cols[k];
    int totalElements = rows * cols;
	int offset = 0;
    for (int m = 0; m < k; m++) {
        offset += d_rows[m] * d_cols[m];
    }
	

	int k=threadIdx.x;
	vector<vector<int>>mat=matrices[k];
	int maxV=range[k];
	vector<int>freqArray(maxV+1,0);
	vector<int>prefixSumArray(maxV+1,0);
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			freqArray[mat[i][j]]++;
		}
	}
	/*updated prefixSumArray Method so if at index i ,
	freqArray[i]!=0 then the output matrix starts the value i 
	from row major index prefixSumArray[i] and continues till 
	it hits another non zero frequency array element*/
	prefixSumArray[0] = 0;
	for (int i = 1; i <= maxV; i++) {
		prefixSumArray[i] = prefixSumArray[i - 1] + freqArray[i-1];
	}
	//to use prefixSumArray and freqArray efficiently using CUDA programming
	//write the updated matxrix to mat
	matrices[k]=mat;
	return;
}

vector<vector<vector<int>>> modify(vector<vector<vector<int>>>& matrices, vector<int>& range){
	int numMatrices = matrices.size();
	// First, flatten matrices into a single contiguous array.
	int totalElements = 0;
    vector<int> rows(numMatrices), cols(numMatrices);
    for (int i = 0; i < numMatrices; i++) {
        int r = matrices[i].size();
        int c = matrices[i][0].size();
        rows[i] = r;
        cols[i] = c;
        totalElements += r * c;
    }

	int* input = new int[totalElements];
    int pos = 0;
    for (int k = 0; k < numMatrices; k++) {
        for (int i = 0; i < rows[k]; i++) {
            for (int j = 0; j < cols[k]; j++) {
                input[pos++] = matrices[k][i][j];
            }
        }
    }

	int *device_input, *device_range, *device_rows, *device_cols;
	hipMalloc(&device_input, totalElements * sizeof(int));
    hipMalloc(&device_range, numMatrices * sizeof(int));
    hipMalloc(&device_rows, numMatrices * sizeof(int));
    hipMalloc(&device_cols, numMatrices * sizeof(int));

	hipMemcpy(device_input, input, numMatrices * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_range, range.data(), numMatrices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_rows, rows.data(), numMatrices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_cols, cols.data(), numMatrices * sizeof(int), hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
    int numBlocks = numMatrices;
	MyKernelFunc<<<numBlocks, threadsPerBlock>>>(device_input, device_range, device_rows, device_cols, numMatrices);
	hipDeviceSynchronize();

	hipMemcpy(input, device_input, totalElements * sizeof(int), hipMemcpyDeviceToHost);
	pos = 0;
    for (int k = 0; k < numMatrices; k++) {
        int r = rows[k];
        int c = cols[k];
        for (int i = 0; i < r; i++) {
            for (int j = 0; j < c; j++) {
                matrices[k][i][j] = input[pos++];
            }
        }
    }

	hipFree(device_input);
    hipFree(device_range);
    hipFree(device_rows);
    hipFree(device_cols);
    delete[] input;
    delete[] output;

	return matrices;
}