#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <stdexcept>
#include "modify.cuh"
#include <cmath>

using namespace std;
int nextPowerOfTwo(int n) {
    int p = 1;
    while (p < n) p *= 2;
    return p;
}
// __global__ void initFreqKernel(int* prefix_global, const int* d_range, int max_padded_size, int numMatrices) {
//     int k = blockIdx.x;
//     if (k >= numMatrices) return;
//     int maxV = d_range[k];
//     int* freqArray = &prefix_global[k * max_padded_size];
//     int tid = threadIdx.x;
//     for (int i = tid; i <= maxV; i += blockDim.x) {
//         freqArray[i] = 0;
//     }
// }
__global__ void countFreqKernel(int* d_input, const int* d_range, const int* d_rows, const int* d_cols, 
                                const int* d_prefix_blocks_pass_0,const int* d_prefix_indices_pass_0, int numMatrices, int* prefix_global, 
                                int max_padded_size, int threadsPerBlock) {
    int block_idx = blockIdx.x;
    int matrix_k = -1;
    for (int i = 0; i < numMatrices; ++i) {
        if (d_prefix_blocks_pass_0[i] <= block_idx && block_idx < d_prefix_blocks_pass_0[i+1]) {
            matrix_k = i;
            break;
        }
    }
    if (matrix_k == -1) return;
    int maxV = d_range[matrix_k];
    int elements = d_rows[matrix_k] * d_cols[matrix_k];
    int startIndexOfFreqArray = d_prefix_indices_pass_0[matrix_k];
    int* freqArray = &prefix_global[startIndexOfFreqArray];
    int offset = 0;
    for (int m = 0; m < matrix_k; m++) {
        offset += d_rows[m] * d_cols[m];
    }
    int* matrix = d_input + offset;
    int blocks_before = block_idx - d_prefix_blocks_pass_0[matrix_k];
    int start = blocks_before * threadsPerBlock;
    // int end = min(start + threadsPerBlock, elements);
    // for (int i = start + threadIdx.x; i < end; i += blockDim.x) {
    int i = start + threadIdx.x;
    int val = matrix[i];
    if (val <= maxV) {
        atomicAdd(&freqArray[val], 1);
    }
    //     break;
    // }
}

__device__ __forceinline__ void inclusiveBlellochScan(int* array, int array_size){
    int tid=threadIdx.x;
    int myVal = array[tid];
    for (int stride = 1; stride < array_size; stride *= 2) {
        int idx = (tid + 1) * 2 * stride - 1;
        if (idx < array_size) {
            array[idx] += array[idx - stride];
        }
        __syncthreads();
    }
    if (tid == 0) {
        array[array_size - 1] = 0;
    }
    __syncthreads();
    for (int stride = array_size / 2; stride > 0; stride /= 2) {
        int idx = (tid + 1) * 2 * stride - 1;
        if (idx < array_size) {
            int temp = array[idx - stride];
            array[idx - stride] = array[idx];
            array[idx] += temp;
        }
        __syncthreads();
    }
    array[tid] += myVal;
    __syncthreads();
}

__global__ void preFixSumKernel(int* d_input, const int* d_range, const int* d_rows, const int* d_cols, 
                                int* prefix_global, const int* d_prefix_blocks_pass_num,const int* d_prefix_indices_pass_num,const int* d_prefix_indices_pass_num_next,int numThreadsPerBlock, int numMatrices, int pass_num) {
    int k = blockIdx.x;
    int matrix_k = -1;
    for (int i = 0; i < numMatrices; ++i) {
        if (d_prefix_blocks_pass_num[i] <= block_idx && block_idx < d_prefix_blocks_pass_num[i+1]) {
            matrix_k = i;
            break;
        }
    }
    if (matrix_k == -1) return;
    if(pass_num==0){
        //just calculate belloch prefix for the array corresponding to that block(inclusive belloch)
        //write the end value of each block at appropriate position for next phase to start
    }else if(pass_num==1){

    }else{//pass_num==2 case
        int array_start_idx=d_prefix_indices_pass_num[matrix_k];
        int* array = &prefix_global[array_start_idx];
        //just inclusive belloch for this array
        int array_size=numThreadsPerBlock;//need to keep it power of 2,fixed as 1024
        
        //propgate it in next kernel
    }
    if (k >= numMatrices) return;
    int maxV = d_range[k];
    int rows = d_rows[k];
    int cols = d_cols[k];
    //int totalElements = rows * cols;
    int n = maxV + 1;
	int p = 1;
    while (p < n) p *= 2;
    int padded_n = p;
    int* prefixSumArray = &prefix_global[k * max_padded_size];
    int tid = threadIdx.x;
    for (int i = tid; i < padded_n; i += blockDim.x) {
        if (i >= n) {
            prefixSumArray[i] = 0;
        }
    }
    __syncthreads();
    for (int stride = 1; stride < padded_n; stride *= 2) {
        for (int i = tid; i < padded_n / (2 * stride); i += blockDim.x) {
            int idx = (i + 1) * 2 * stride - 1;
            if (idx < padded_n) {
                prefixSumArray[idx] += prefixSumArray[idx - stride];
            }
        }
        __syncthreads();
    }
    if (tid == 0) {
        prefixSumArray[padded_n - 1] = 0;
    }
    __syncthreads();
    for (int stride = padded_n / 2; stride > 0; stride /= 2) {
        for (int i = tid; i < padded_n / (2 * stride); i += blockDim.x) {
            int idx = (i + 1) * 2 * stride - 1;
            if (idx < padded_n) {
                int temp = prefixSumArray[idx - stride];
                prefixSumArray[idx - stride] = prefixSumArray[idx];
                prefixSumArray[idx] += temp;
            }
        }
        __syncthreads();
    }
    // int offset = 0;
    // for (int m = 0; m < k; m++) {
    //     offset += d_rows[m] * d_cols[m];
    // }
    // int* matrix = d_input + offset;
    // for (int val = tid; val <= maxV; val += blockDim.x) {
    //     int start = prefixSumArray[val];
    //     int end = (val == maxV) ? totalElements : prefixSumArray[val + 1];
    //     if (start >= totalElements || end > totalElements) continue;
    //     for (int pos = start; pos < end; pos++) {
    //         matrix[pos] = val;
    //     }
    // }
}

__global__ void writeBackKernel(int* d_input, const int* d_range, const int* d_rows, const int* d_cols, const int* d_prefix_blocks_write, int max_padded_size, int numMatrices, int* prefix_global) {
    int block_idx = blockIdx.x;
    int matrix_k = -1;
    for (int i = 0; i < numMatrices; ++i) {
        if (d_prefix_blocks_write[i] <= block_idx && block_idx < d_prefix_blocks_write[i+1]) {
            matrix_k = i;
            break;
        }
    }
    if (matrix_k == -1) return;

    int maxV = d_range[matrix_k];
    int elements = d_rows[matrix_k] * d_cols[matrix_k];
    int* prefixArr = &prefix_global[matrix_k * max_padded_size];

    // for (int m = 0; m < matrix_k; m++) {
    //     offset += d_rows[m] * d_cols[m];
    // }
    //int* matrix = d_input + offset;
    //int blocks_before = block_idx - d_prefix_blocks[matrix_k];
    //int start = blocks_before * threadsPerBlock;
    //int end = min(start + threadsPerBlock, elements);
    // for (int i = start + threadIdx.x; i < end; i += blockDim.x) {
    //     int val = matrix[i];
    //     if (val <= maxV) {
    //         atomicAdd(&freqArray[val], 1);
    //     }
    // }
    
    int block_offset = block_idx - d_prefix_blocks_write[matrix_k];
    int val_start = block_offset * blockDim.x;
    int val_end = min(val_start + blockDim.x, maxV + 1);
    
    int input_offset = 0;
    for (int m = 0; m < matrix_k; m++){
        input_offset += d_rows[m] * d_cols[m];
    }
    for (int val = val_start + threadIdx.x; val < val_end; val += blockDim.x) {
        int start = prefixArr[val];
        int end = (val == maxV) ? elements : prefixArr[val + 1];
        
        int* matrix = d_input + input_offset;
        for (int pos = start; pos < end; ++pos)
            matrix[pos] = val;
    }
}
struct CudaPtrGuard {
    void** ptr;
    explicit CudaPtrGuard(void** p) : ptr(p) {}
    ~CudaPtrGuard() { 
        if (ptr && *ptr) {
            hipFree(*ptr); 
            *ptr = nullptr;
        }
    }
};
void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        cerr << "CUDA Error (" << msg << "): " << hipGetErrorString(err) << endl;
        throw runtime_error(hipGetErrorString(err));
    }
}
vector<vector<vector<int>>> modify(vector<vector<vector<int>>>& matrices, vector<int>& range) {
    int* host_input = nullptr;
    vector<int> rows, cols, prefix_indices_pass_0, prefix_indices_pass_1, prefix_indices_pass_2,prefix_blocks_pass_0,prefix_blocks_pass_1,prefix_blocks_pass_2;
    int *d_input = nullptr, *d_range = nullptr;
    int *d_rows = nullptr;
    int *d_cols = nullptr;
    int *d_prefix_indices_pass_0 = nullptr, *d_prefix_indices_pass_1 = nullptr, *d_prefix_indices_pass_2 = nullptr;
    int *prefix_global = nullptr;
    int*d_prefix_blocks_pass_0 = nullptr,*d_prefix_blocks_pass_1 = nullptr,*d_prefix_blocks_pass_2 = nullptr;
    int numThreads = 1024;
    try {
        const int numMatrices = matrices.size();
        rows.resize(numMatrices);
        cols.resize(numMatrices);
        prefix_indices_pass_0.resize(numMatrices);
        prefix_indices_pass_1.resize(numMatrices);
        prefix_indices_pass_2.resize(numMatrices);
        prefix_blocks_pass_0.resize(numMatrices+1);
        prefix_blocks_pass_1.resize(numMatrices+1);
        prefix_blocks_pass_2.resize(numMatrices+1);
        int Elements_0 = 0;
        int Elements_1 = 0;
        int Elements_2 = 0;
        int Blocks_0 = 0;
        int Blocks_1 = 0;
        int Blocks_2 = 0;
        int totalElements = 0;//total elements in the input
        int totalElementsInPrefix = 0;
        for (int i = 0; i < numMatrices; i++) {
            if (matrices[i].empty() || matrices[i][0].empty()) {
                throw runtime_error("Empty matrix detected");
            }
            rows[i] = matrices[i].size();
            cols[i] = matrices[i][0].size();
            prefix_indices_pass_0[i] = Elements_0+Elements_1+Elements_2;
            if(i>0){
                prefix_indices_pass_0[i]+=prefix_indices_pass_0[i-1];
            }
            Elements_0 = rows[i] * cols[i];
            Blocks_0 = static_cast<int>(std::ceil(static_cast<double>(Elements_0) / numThreads));
            prefix_blocks_pass_0[i+1]=prefix_blocks_pass_0[i]+Blocks_0;
            prefix_indices_pass_1[i] = Elements_0+Elements_1+Elements_2;
            if(i>0){
                prefix_indices_pass_1[i]+=prefix_indices_pass_1[i-1];
            }
            Elements_1 = Blocks_0*numThreads;
            Blocks_1 = static_cast<int>(std::ceil(static_cast<double>(Elements_1) / numThreads));
            prefix_blocks_pass_1[i+1]=prefix_blocks_pass_1[i]+Blocks_1;
            prefix_indices_pass_2[i] = Elements_0+Elements_1+Elements_2;
            if(i>0){
                prefix_indices_pass_2[i]+=prefix_indices_pass_2[i-1];
            }
            Elements_2 = Blocks_1*numThreads;
            Blocks_1 = static_cast<int>(std::ceil(static_cast<double>(Elements_2) / numThreads));
            prefix_blocks_pass_2[i+1]=prefix_blocks_pass_2[i]+Blocks_2;
            totalElementsInPrefix+=Elements_0+Elements_1+Elements_2;
            totalElements+=1;
        }
        int pos=0;
        host_input = new int[totalElements];
        for (int k = 0; k < numMatrices; k++) {
            for (const auto& row : matrices[k]) {
                for (int val : row) {
                    host_input[pos++] = val;
                }
            }
        }
        checkCuda(hipMalloc(&d_input, totalElements * sizeof(int)), "d_input alloc");
        CudaPtrGuard guard_d_input(reinterpret_cast<void**>(&d_input));
        checkCuda(hipMalloc(&d_range, numMatrices * sizeof(int)), "d_range alloc");
        CudaPtrGuard guard_d_range(reinterpret_cast<void**>(&d_range));
        checkCuda(hipMalloc(&d_rows, numMatrices * sizeof(int)), "d_rows alloc");
        CudaPtrGuard guard_d_rows(reinterpret_cast<void**>(&d_rows));
        checkCuda(hipMalloc(&d_cols, numMatrices * sizeof(int)), "d_cols alloc");
        CudaPtrGuard guard_d_cols(reinterpret_cast<void**>(&d_cols));
        checkCuda(hipMalloc(&d_prefix_indices_pass_0, numMatrices * sizeof(int)), "d_prefix_indices_pass_0 alloc");
        CudaPtrGuard guard_d_cols(reinterpret_cast<void**>(&d_prefix_indices_pass_0));
        checkCuda(hipMalloc(&d_prefix_indices_pass_1, numMatrices * sizeof(int)), "d_prefix_indices_pass_1 alloc");
        CudaPtrGuard guard_d_cols(reinterpret_cast<void**>(&d_prefix_indices_pass_1));
        checkCuda(hipMalloc(&d_prefix_indices_pass_2, numMatrices * sizeof(int)), "d_prefix_indices_pass_2 alloc");
        CudaPtrGuard guard_d_cols(reinterpret_cast<void**>(&d_prefix_indices_pass_2));
        checkCuda(hipMalloc(&d_prefix_blocks_pass_0, numMatrices * sizeof(int)), "d_prefix_blocks_pass_0 alloc");
        CudaPtrGuard guard_d_cols(reinterpret_cast<void**>(&d_prefix_blocks_pass_0));
        checkCuda(hipMalloc(&d_prefix_blocks_pass_1, numMatrices * sizeof(int)), "d_prefix_blocks_pass_1 alloc");
        CudaPtrGuard guard_d_cols(reinterpret_cast<void**>(&d_prefix_blocks_pass_1));
        checkCuda(hipMalloc(&d_prefix_blocks_pass_2, numMatrices * sizeof(int)), "d_prefix_blocks_pass_2 alloc");
        CudaPtrGuard guard_d_cols(reinterpret_cast<void**>(&d_prefix_blocks_pass_2));
        checkCuda(hipMalloc(&prefix_global, totalElementsInPrefix * sizeof(int)),"prefix_global alloc");
        CudaPtrGuard guard_prefix_global(reinterpret_cast<void**>(&prefix_global));
        checkCuda(hipMemcpy(d_input, host_input, totalElements * sizeof(int), hipMemcpyHostToDevice), "d_input copy");
        checkCuda(hipMemcpy(d_range, range.data(), numMatrices * sizeof(int), hipMemcpyHostToDevice), "d_range copy");
        checkCuda(hipMemcpy(d_rows, rows.data(), numMatrices * sizeof(int), hipMemcpyHostToDevice), "d_rows copy");
        checkCuda(hipMemcpy(d_cols, cols.data(), numMatrices * sizeof(int), hipMemcpyHostToDevice), "d_cols copy");
        checkCuda(hipMemcpy(d_prefix_indices_pass_0, prefix_indices_pass_0.data(), numMatrices * sizeof(int), hipMemcpyHostToDevice), "d_prefix_indices_pass_0 copy");
        checkCuda(hipMemcpy(d_prefix_indices_pass_1, prefix_indices_pass_1.data(), numMatrices * sizeof(int), hipMemcpyHostToDevice), "d_prefix_indices_pass_1 copy");
        checkCuda(hipMemcpy(d_prefix_indices_pass_2, prefix_indices_pass_2.data(), numMatrices * sizeof(int), hipMemcpyHostToDevice), "d_prefix_indices_pass_2 copy");
        checkCuda(hipMemcpy(d_prefix_blocks_pass_0, prefix_blocks_pass_0.data(), numMatrices * sizeof(int), hipMemcpyHostToDevice), "d_prefix_blocks_pass_0 copy");
        checkCuda(hipMemcpy(d_prefix_blocks_pass_1, prefix_blocks_pass_1.data(), numMatrices * sizeof(int), hipMemcpyHostToDevice), "d_prefix_blocks_pass_1 copy");
        checkCuda(hipMemcpy(d_prefix_blocks_pass_2, prefix_blocks_pass_2.data(), numMatrices * sizeof(int), hipMemcpyHostToDevice), "d_prefix_blocks_pass_2 copy");

        // initFreqKernel<<<numMatrices, 1024>>>(prefix_global, d_range, max_padded_size, numMatrices);
        // checkCuda(hipGetLastError(), "initFreqKernel launch");
        // checkCuda(hipDeviceSynchronize(), "initFreqKernel sync");
        countFreqKernel<<<prefix_blocks[numMatrices], 1024>>>(d_input, d_range, d_rows, d_cols, d_prefix_blocks, numMatrices, prefix_global, max_padded_size, 1024);
        checkCuda(hipGetLastError(), "countFreqKernel launch");
        checkCuda(hipDeviceSynchronize(), "countFreqKernel sync");
        preFixSumKernel<<<numMatrices, 1024>>>(d_input, d_range, d_rows, d_cols, numMatrices, prefix_global, max_padded_size);
        checkCuda(hipGetLastError(), "preFixSumKernel launch");
        checkCuda(hipDeviceSynchronize(), "preFixSumKernel sync");
        vector<int> prefix_blocks_write(numMatrices + 1, 0);
        for (int i = 0; i < numMatrices; ++i) {
            int blocks = (range[i] + 1 + 1023) / 1024;
            prefix_blocks_write[i+1] = prefix_blocks_write[i] + blocks;
        }
        checkCuda(hipMalloc(&d_prefix_blocks_write, (numMatrices+1)*sizeof(int)), "d_prefix_blocks_write alloc");
        checkCuda(hipMemcpy(d_prefix_blocks_write, prefix_blocks_write.data(), (numMatrices+1)*sizeof(int), hipMemcpyHostToDevice), "d_prefix_blocks_write copy");
        writeBackKernel<<<prefix_blocks_write[numMatrices], 1024>>>(d_input, d_range, d_rows, d_cols, d_prefix_blocks_write, max_padded_size, numMatrices, prefix_global);
        checkCuda(hipGetLastError(), "writeBackSumKernel launch");
        checkCuda(hipDeviceSynchronize(), "writeBackSumKernel sync");
        checkCuda(hipMemcpy(host_input, d_input, totalElements * sizeof(int), hipMemcpyDeviceToHost), "results copy");
        pos = 0;
        for (int k = 0; k < numMatrices; k++) {
            const int r = rows[k], c = cols[k];
            for (int i = 0; i < r; i++) {
                for (int j = 0; j < c; j++) {
                    matrices[k][i][j] = host_input[pos++];
                }
            }
        }
        delete[] host_input;
        host_input = nullptr;
        return matrices;
    } catch (...) {
        if (host_input) delete[] host_input;
        if (d_input) hipFree(d_input);
        if (d_range) hipFree(d_range);
        if (d_rows) hipFree(d_rows);
        if (d_cols) hipFree(d_cols);
        if (prefix_global) hipFree(prefix_global);
        if (d_prefix_blocks) hipFree(d_prefix_blocks);
        hipDeviceReset();
        throw;
    }
}