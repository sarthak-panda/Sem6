#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <stdexcept>
#include "modify.cuh"

using namespace std;

// Helper function to find the next power of two
int nextPowerOfTwo(int n) {
    int p = 1;
    while (p < n) p *= 2;
    return p;
}

// Kernel to initialize frequency arrays
__global__ void initFreqKernel(int* freq_global, const int* d_range, int max_freq_size, int numMatrices) {
    int k = blockIdx.x;
    if (k >= numMatrices) return;
    int maxV = d_range[k];
    int* freqArray = &freq_global[k * max_freq_size];
    int tid = threadIdx.x;
    for (int i = tid; i <= maxV; i += blockDim.x) {
        freqArray[i] = 0;
    }
}

// Kernel to count frequencies using multiple blocks per matrix
__global__ void countFreqKernel(int* d_input, const int* d_range, const int* d_rows, const int* d_cols, const int* d_prefix_blocks, int numMatrices, int* freq_global, int max_freq_size, int threadsPerBlock) {
    int block_idx = blockIdx.x;
    int matrix_k = -1;
    for (int i = 0; i < numMatrices; ++i) {
        if (d_prefix_blocks[i] <= block_idx && block_idx < d_prefix_blocks[i+1]) {
            matrix_k = i;
            break;
        }
    }
    if (matrix_k == -1) return;

    int maxV = d_range[matrix_k];
    int elements = d_rows[matrix_k] * d_cols[matrix_k];
    int* freqArray = &freq_global[matrix_k * max_freq_size];

    int offset = 0;
    for (int m = 0; m < matrix_k; m++) {
        offset += d_rows[m] * d_cols[m];
    }
    int* matrix = d_input + offset;

    int blocks_before = block_idx - d_prefix_blocks[matrix_k];
    int start = blocks_before * threadsPerBlock;
    int end = min(start + threadsPerBlock, elements);

    for (int i = start + threadIdx.x; i < end; i += blockDim.x) {
        int val = matrix[i];
        if (val <= maxV) {
            atomicAdd(&freqArray[val], 1);
        }
    }
}

// Kernel to compute prefix sum and write back values
__global__ void writeBackKernel(int* d_input, const int* d_range, const int* d_rows, const int* d_cols, int numMatrices, int* freq_global, int* prefix_global, int max_freq_size, int max_padded_size) {
    int k = blockIdx.x;
    if (k >= numMatrices) return;

    int maxV = d_range[k];
    int rows = d_rows[k];
    int cols = d_cols[k];
    int totalElements = rows * cols;
    int n = maxV + 1;
    int p = 1;
    while (p < n) p *= 2;
    int padded_n = p;
    int* freqArray = &freq_global[k * max_freq_size];
    int* prefixSumArray = &prefix_global[k * max_padded_size];
    int tid = threadIdx.x;

    // Copy freqArray to prefixSumArray and pad
    for (int i = tid; i < padded_n; i += blockDim.x) {
        if (i < n) {
            prefixSumArray[i] = freqArray[i];
        } else {
            prefixSumArray[i] = 0;
        }
    }
    __syncthreads();

    // Blelloch scan
    // Up-sweep
    for (int stride = 1; stride < padded_n; stride *= 2) {
        for (int i = tid; i < padded_n / (2 * stride); i += blockDim.x) {
            int idx = (i + 1) * 2 * stride - 1;
            if (idx < padded_n) {
                prefixSumArray[idx] += prefixSumArray[idx - stride];
            }
        }
        __syncthreads();
    }

    // Down-sweep
    if (tid == 0) {
        prefixSumArray[padded_n - 1] = 0;
    }
    __syncthreads();

    for (int stride = padded_n / 2; stride > 0; stride /= 2) {
        for (int i = tid; i < padded_n / (2 * stride); i += blockDim.x) {
            int idx = (i + 1) * 2 * stride - 1;
            if (idx < padded_n) {
                int temp = prefixSumArray[idx - stride];
                prefixSumArray[idx - stride] = prefixSumArray[idx];
                prefixSumArray[idx] += temp;
            }
        }
        __syncthreads();
    }

    // Write back
    int offset = 0;
    for (int m = 0; m < k; m++) {
        offset += d_rows[m] * d_cols[m];
    }
    int* matrix = d_input + offset;

    for (int val = tid; val <= maxV; val += blockDim.x) {
        int start = prefixSumArray[val];
        int end = (val == maxV) ? totalElements : prefixSumArray[val + 1];
        for (int pos = start; pos < end; pos++) {
            matrix[pos] = val;
        }
    }
}


// RAII guard for CUDA pointers
struct CudaPtrGuard {
    void** ptr;
    explicit CudaPtrGuard(void** p) : ptr(p) {}
    ~CudaPtrGuard() { 
        if (*ptr) {
            hipFree(*ptr); 
            *ptr = nullptr;
        }
    }
};

// Error checking wrapper
void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        cerr << "CUDA Error (" << msg << "): " 
             << hipGetErrorString(err) << endl;
        throw runtime_error(hipGetErrorString(err));
    }
}

vector<vector<vector<int>>> modify(vector<vector<vector<int>>>& matrices, 
                                  vector<int>& range) {
    // Host memory management
    int* host_input = nullptr;
    vector<int> rows, cols;
    
    // Device pointers (all initialized to nullptr)
    int *d_input = nullptr, *d_range = nullptr, *d_rows = nullptr;
    int *d_cols = nullptr, *freq_global = nullptr;
    int *prefix_global = nullptr, *d_prefix_blocks = nullptr;

    try {
        // ====================== Host Setup ======================
        const int numMatrices = matrices.size();
        rows.resize(numMatrices);
        cols.resize(numMatrices);
        int totalElements = 0;

        // Validate input matrices
        for (int i = 0; i < numMatrices; i++) {
            if (matrices[i].empty() || matrices[i][0].empty()) {
                throw runtime_error("Empty matrix detected");
            }
            rows[i] = matrices[i].size();
            cols[i] = matrices[i][0].size();
            totalElements += rows[i] * cols[i];
        }

        // Flatten input matrices
        host_input = new int[totalElements];
        int pos = 0;
        for (int k = 0; k < numMatrices; k++) {
            for (const auto& row : matrices[k]) {
                for (int val : row) {
                    host_input[pos++] = val;
                }
            }
        }

        // ====================== Device Allocation ======================
        // Each allocation is immediately guarded
        checkCuda(hipMalloc(&d_input, totalElements * sizeof(int)), "d_input alloc");
        CudaPtrGuard guard_d_input(reinterpret_cast<void**>(&d_input));

        checkCuda(hipMalloc(&d_range, numMatrices * sizeof(int)), "d_range alloc");
        CudaPtrGuard guard_d_range(reinterpret_cast<void**>(&d_range));

        checkCuda(hipMalloc(&d_rows, numMatrices * sizeof(int)), "d_rows alloc");
        CudaPtrGuard guard_d_rows(reinterpret_cast<void**>(&d_rows));

        checkCuda(hipMalloc(&d_cols, numMatrices * sizeof(int)), "d_cols alloc");
        CudaPtrGuard guard_d_cols(reinterpret_cast<void**>(&d_cols));

        const int max_range = *max_element(range.begin(), range.end());
        const int max_freq_size = max_range + 1;
        const int max_padded_size = nextPowerOfTwo(max_freq_size);

        checkCuda(hipMalloc(&freq_global, numMatrices * max_freq_size * sizeof(int)), 
                "freq_global alloc");
        CudaPtrGuard guard_freq_global(reinterpret_cast<void**>(&freq_global));

        checkCuda(hipMalloc(&prefix_global, numMatrices * max_padded_size * sizeof(int)), 
                "prefix_global alloc");
        CudaPtrGuard guard_prefix_global(reinterpret_cast<void**>(&prefix_global));

        // ====================== Data Transfers ======================
        checkCuda(hipMemcpy(d_input, host_input, totalElements * sizeof(int), 
                           hipMemcpyHostToDevice), "d_input copy");
        checkCuda(hipMemcpy(d_range, range.data(), numMatrices * sizeof(int),
                           hipMemcpyHostToDevice), "d_range copy");
        checkCuda(hipMemcpy(d_rows, rows.data(), numMatrices * sizeof(int),
                           hipMemcpyHostToDevice), "d_rows copy");
        checkCuda(hipMemcpy(d_cols, cols.data(), numMatrices * sizeof(int),
                           hipMemcpyHostToDevice), "d_cols copy");

        // ====================== Kernel Setup ======================
        vector<int> prefix_blocks(numMatrices + 1, 0);
        for (int i = 0; i < numMatrices; ++i) {
            const int elements = rows[i] * cols[i];
            prefix_blocks[i+1] = prefix_blocks[i] + (elements + 1023)/1024;
        }

        checkCuda(hipMalloc(&d_prefix_blocks, (numMatrices + 1) * sizeof(int)), 
                "d_prefix_blocks alloc");
        CudaPtrGuard guard_prefix_blocks(reinterpret_cast<void**>(&d_prefix_blocks));
        checkCuda(hipMemcpy(d_prefix_blocks, prefix_blocks.data(), 
                           (numMatrices + 1) * sizeof(int), hipMemcpyHostToDevice),
                           "d_prefix_blocks copy");

        // ====================== Kernel Execution ======================
        // Initialize frequency arrays
        initFreqKernel<<<numMatrices, 1024>>>(freq_global, d_range, max_freq_size, numMatrices);
        checkCuda(hipGetLastError(), "initFreqKernel launch");
        checkCuda(hipDeviceSynchronize(), "initFreqKernel sync");

        // Count frequencies
        countFreqKernel<<<prefix_blocks[numMatrices], 1024>>>(d_input, d_range, d_rows, d_cols,
                                                             d_prefix_blocks, numMatrices,
                                                             freq_global, max_freq_size, 1024);
        checkCuda(hipGetLastError(), "countFreqKernel launch");
        checkCuda(hipDeviceSynchronize(), "countFreqKernel sync");

        // Compute prefix sums and write back
        writeBackKernel<<<numMatrices, 1024>>>(d_input, d_range, d_rows, d_cols, numMatrices,
                                             freq_global, prefix_global, max_freq_size, 
                                             max_padded_size);
        checkCuda(hipGetLastError(), "writeBackKernel launch");
        checkCuda(hipDeviceSynchronize(), "writeBackKernel sync");

        // ====================== Retrieve Results ======================
        checkCuda(hipMemcpy(host_input, d_input, totalElements * sizeof(int),
                           hipMemcpyDeviceToHost), "results copy");

        // Update matrices
        pos = 0;
        for (int k = 0; k < numMatrices; k++) {
            const int r = rows[k], c = cols[k];
            for (int i = 0; i < r; i++) {
                for (int j = 0; j < c; j++) {
                    matrices[k][i][j] = host_input[pos++];
                }
            }
        }

        // ====================== Explicit Cleanup (Optional) ======================
        // Guards will auto-free, but explicit cleanup ensures early release
        hipFree(d_input); d_input = nullptr;
        hipFree(d_range); d_range = nullptr;
        hipFree(d_rows); d_rows = nullptr;
        hipFree(d_cols); d_cols = nullptr;
        hipFree(freq_global); freq_global = nullptr;
        hipFree(prefix_global); prefix_global = nullptr;
        hipFree(d_prefix_blocks); d_prefix_blocks = nullptr;

        delete[] host_input;
        host_input = nullptr;

        return matrices;

    } catch (...) {
        // Cleanup host memory if error occurred before normal deletion
        if (host_input) {
            delete[] host_input;
            host_input = nullptr;
        }

		// Explicit device cleanup (complementary to RAII)
		if (d_input) hipFree(d_input);
		if (d_range) hipFree(d_range);
		if (d_rows) hipFree(d_rows);
		if (d_cols) hipFree(d_cols);
		if (freq_global) hipFree(freq_global);
		if (prefix_global) hipFree(prefix_global);
		if (d_prefix_blocks) hipFree(d_prefix_blocks);

		// Nuclear option: Reset GPU to clean up any leaked context
		hipError_t reset_err = hipDeviceReset();
		if (reset_err != hipSuccess) {
			cerr << "hipDeviceReset() failed: " 
				 << hipGetErrorString(reset_err) << endl;
		}
        
        // Re-throw to notify caller
        throw;
    }
}